#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

#define N 2500

__global__ void vector_add(float *out, float *a, float *b, int n)
{
    for (int i = 0; i < n; i++)
    {
        out[i] = a[i] + b[i];
    }
}
int main()
{
    float *a, *b, *out;
    float *d_a, *d_b;

    a = (float *)malloc(sizeof(float) * N);

    // Allocate device memory for a
    hipMalloc((void **)&d_a, sizeof(float) * N);

    auto startTime = std::chrono::high_resolution_clock::now();

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    auto endTime = std::chrono::high_resolution_clock::now();

    // Calculate the elapsed time in nanoseconds
    auto elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime).count();

    // Print the results
    std::cout << "Elapsed Time (nanoseconds): " << elapsedTime << std::endl;

    vector_add<<<1, 1>>>(out, d_a, b, N);

    // Cleanup after kernel execution
    hipFree(d_a);
    free(a);
}